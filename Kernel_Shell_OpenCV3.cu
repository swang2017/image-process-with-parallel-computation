#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <stdio.h>

#include<ctime>


using namespace cv;
using namespace std;

void Thresholding(const Mat& input, Mat& output, unsigned char th)
{
	// TODO: Add your CPU side serial code to perform thresholding here
}




//CUDA function protorype. It takes output image and input image and a threshold value
hipError_t  performWithCuda(Mat&, const Mat&, unsigned char th);

// CUDA GPU Kernel 
__global__ void gpuThreshold(unsigned char *b, const unsigned char * a, unsigned char th, unsigned int r, unsigned int c)
{
	// TODO: 
	//  1- calculate the index of the pointers based on pixel location for each thread
	//	2- perform the thresholding

}

int main()
{
	unsigned char threshold = 128;		// This is a threshold value, you can change this value
	hipError_t cudaStatus;			// This is the hipError_t code that your functions may return to troubleshoot



	//	TODO: 
	//	1- Read the input gray-scale image with imread
	//		1-1- if image has no data show an error message
	//		1-2- if iamge has data
	//			1-2-1- create an image for the CPU output, and one for the GPU output
	//			1-2-2- call your CPU  side code to threshold the image (pass the input image and the cpu output image and the threshold)
	//			1-2-3- call the performWithCuda function to create gpu pointers, copy data from host to device, invoke kernel 
	//						and copy results back to host (refer to the above function prototype on line 23 for reference.)
	//			1-2-4- Use imshow to show the input image, the CPU output and the GPU output. Note: CPU and GPU outputs should look alike.
	//		1-3- use cvWaitKey(0); to pause.





	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	//		DO NOT CHANGE THE FOLLOWING!
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


	cvWaitKey(0);


	return 0;
}

// Helper function for using CUDA to perform image thresholding in parallel. Takes as input the thresholded image (bwImage), the input image (input), and the threshold value.
hipError_t performWithCuda(Mat &bwImage, const Mat &input, unsigned char threshold)
{
	unsigned char *dev_ptrout, *dev_ptrin;	// these are the gpu side ouput and input pointers

	hipError_t cudaStatus;



	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	// Allocate GPU buffers for the buffers (one input, one output)    .
	// TODO: add your code here to allocate the input pointer on the device. Note the size of the pointer in hipMalloc

	// TODO: add your code here to allocate the outpu pointer on the device. Note the size of the pointer in hipMalloc


	// Copy input data from host memory to GPU buffers.
	// TODO: Add your code here. Use hipMemcpy


	// TODO: Launch a kernel on the GPU with one thread for each element. use <<< grid_size (or number of blocks), block_size(or number of threads) >>>

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// TODO: Copy output data from GPU buffer to host memory. use hipMemcpy

Error:
	hipFree(dev_ptrin);
	hipFree(dev_ptrout);

	return cudaStatus;
}